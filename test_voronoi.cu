#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cassert>
#include <cstdlib>
#include <cstdio>

#include "params.h"
#include "voronoi.h"
#include "stopwatch.h"

void get_bbox(const std::vector<float>& vertices, float& xmin, float& ymin, float& zmin, float& xmax, float& ymax, float& zmax) {
    int nb_v = vertices.size() / 3;
    xmin = xmax = vertices[0];
    ymin = ymax = vertices[1];
    zmin = zmax = vertices[2];
    for (int i = 1; i < nb_v; ++i)
    {
        xmin = std::min(xmin, vertices[3 * i]);
        ymin = std::min(ymin, vertices[3 * i + 1]);
        zmin = std::min(zmin, vertices[3 * i + 2]);
        xmax = std::max(xmax, vertices[3 * i]);
        ymax = std::max(ymax, vertices[3 * i + 1]);
        zmax = std::max(zmax, vertices[3 * i + 2]);
    }
    float d = xmax - xmin;
    d = std::max(d, ymax - ymin);
    d = std::max(d, zmax - zmin);
    d = 0.001f * d;
    xmin -= d;
    ymin -= d;
    zmin -= d;
    xmax += d;
    ymax += d;
    zmax += d;
}

bool load_tet(
    const std::string& filename,
    std::vector<float>& vertices,
    std::vector<int>& indices,
    bool normalize = true
)
{
    std::string s;
    int n_vertex, n_tet, temp;

    std::ifstream input(filename);
    if (input.fail())
        return false;

    std::string ext = filename.substr(filename.find_last_of('.') + 1);
    if (ext == "tet")
    {
        input >> n_vertex;
        std::getline(input, s);
        input >> n_tet;
        std::getline(input, s);

        vertices.resize(3 * n_vertex);
        indices.resize(n_tet << 2);

        for (int i = 0; i < n_vertex; ++i)
            input >> vertices[3 * i] >> vertices[3 * i + 1] >> vertices[3 * i + 2];

        for (int i = 0; i < n_tet; ++i)
        {
            input >> temp >> indices[(i << 2)] >> indices[(i << 2) + 1] >> indices[(i << 2) + 2] >> indices[(i << 2) + 3];
            assert(temp == 4);
        }
    }
    else if (ext == "vtk")
    {
        for (int i = 0; i < 4; ++i)
            std::getline(input, s); // skip first 4 lines
        
        input >> s >> n_vertex >> s;
        vertices.resize(3 * n_vertex);
        for (int i = 0; i < n_vertex; ++i)
            input >> vertices[3 * i] >> vertices[3 * i + 1] >> vertices[3 * i + 2];

        input >> s >> n_tet >> s;
        indices.resize(n_tet << 2);
        for (int i = 0; i < n_tet; ++i)
        {
            input >> temp >> indices[(i << 2)] >> indices[(i << 2) + 1] >> indices[(i << 2) + 2] >> indices[(i << 2) + 3];
            assert(temp == 4);
            for (int j = 0; j < 4; ++j)
                --indices[(i << 2) + j];
        }
    }
    else
    {
        input.close();
        return false;
    }


    input.close();

    float xmin, ymin, zmin, xmax, ymax, zmax;
    get_bbox(vertices, xmin, ymin, zmin, xmax, ymax, zmax);

    if (normalize) // normalize vertices between [0,1000]^3
    {
        float maxside = std::max(std::max(xmax - xmin, ymax - ymin), zmax - zmin);
#pragma omp parallel for
        for (int i = 0; i < n_vertex; i++)
        {
            vertices[3 * i] = 1000.f * (vertices[3 * i] - xmin) / maxside;
            vertices[3 * i + 1] = 1000.f * (vertices[3 * i + 1] - ymin) / maxside;
            vertices[3 * i + 2] = 1000.f * (vertices[3 * i + 2] - zmin) / maxside;
        }
        get_bbox(vertices, xmin, ymin, zmin, xmax, ymax, zmax);
        std::cerr << "bbox [" << xmin << ":" << xmax << "], [" << ymin << ":" << ymax << "], [" << zmin << ":" << zmax << "]" << std::endl;
    }

    return true;
}

void drop_xyz_file(const bool site_is_transposed, const std::vector<float>& site, const int n_site, const char *filename) {
    std::fstream file;
    file.open(filename, std::ios_base::out);
    file << n_site << std::endl;
    for(int i = 0; i < n_site; i++)
        if (site_is_transposed)
            file << site[i] << "  " << site[i + n_site] << "  " << site[i + (n_site << 1)] << std::endl;
        else
            file << site[3 * i] << "  " << site[3 * i + 1] << "  " << site[3 * i + 2] << std::endl;
    file.close();
}

void load_xyz_file(bool& site_is_transposed, std::vector<float>& site, int& n_site, const char* filename)
{
    std::ifstream file(filename);
    file >> n_site;
    site_is_transposed = n_site < KNEARESTS_MIN_N; // use knn_cuda_global_dev if true, else knearests
    site.resize(n_site * 3);
    for (int i = 0; i < n_site; ++i)
        if (site_is_transposed)
            file >> site[i] >> site[i + n_site] >> site[i + (n_site << 1)];
        else
            file >> site[3 * i] >> site[3 * i + 1] >> site[3 * i + 2];
    file.close();
}

void printDevProp() {
    
    int devCount; // Number of CUDA devices
    hipError_t err = hipGetDeviceCount(&devCount);
    if (err != hipSuccess) {
        std::cerr << "Failed to initialize CUDA / failed to count CUDA devices (error code << "
		  << hipGetErrorString(err) << ")! [file: " << __FILE__ << ", line: " <<  __LINE__ << "]" << std::endl;
        exit(1);
    }
    
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    // Iterate through devices
    for (int i=0; i<devCount; ++i) {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printf("Major revision number:         %d\n",  devProp.major);
        printf("Minor revision number:         %d\n",  devProp.minor);
        printf("Name:                          %s\n",  devProp.name);
        printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
        printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
        printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
        printf("Warp size:                     %d\n",  devProp.warpSize);
        printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
        printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
        for (int i = 0; i < 3; ++i)
            printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
        for (int i = 0; i < 3; ++i)
            printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
        printf("Clock rate:                    %d\n",  devProp.clockRate);
        printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
        printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
        printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
        printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
        printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    }
}

int main(int argc, char** argv) {
    printDevProp();
    if (4 > argc)
    {
        std::cerr << "Usage: " << argv[0] << " <tet_mesh.tet/vtk> <sites_file.xyz> <nb_iter> <k (optional)> (e.g.: " << argv[0] << " ../data/joint.tet ../data/joint.xyz 120 0)" << std::endl;
        return 1;
    }
    int* initptr = nullptr;
    hipError_t err = hipMalloc(&initptr, sizeof(int)); // unused memory, needed for initialize the GPU before time measurements
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate (error code << " << hipGetErrorString(err) << ")! [file: " << __FILE__ << ", line: " << __LINE__ << "]" << std::endl;
        return 1;
    }

    std::vector<float> vertices;
    std::vector<int> indices;

    if (!load_tet(argv[1], vertices, indices))
    {
        std::cerr << argv[1] << ": could not load file" << std::endl;
        return 1;
    }

    int n_site;
    bool site_is_transposed;
    std::vector<float> site;
    load_xyz_file(site_is_transposed, site, n_site, argv[2]);

    if (5 == argc)
        compute_clipped_voro_diagram_GPU(vertices, indices, site, n_site, site_is_transposed, atoi(argv[3]), atoi(argv[4]));
    else
        compute_clipped_voro_diagram_GPU(vertices, indices, site, n_site, site_is_transposed, atoi(argv[3]));

    drop_xyz_file(site_is_transposed, site, n_site, "out.xyz");

    hipFree(initptr);
    return 0;
}
